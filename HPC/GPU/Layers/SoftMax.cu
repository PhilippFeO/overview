#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

// Zum Berechnen des zeilenweisen Extrema
#include <thrust/extrema.h>

using std::cerr;
using std::cout;
using std::endl;

#include "../Network_Config.h"


__global__ void forwardKernel_SoftMax(float* input, float* output){
	const int row = blockIdx.x;
	const int column = threadIdx.x;

	if (row < BATCH_SIZE) {
		__shared__ float sumOfRow;
		sumOfRow = 0;
		__syncthreads();

		if (column < NEURONS_LAYER3) {
			float t = expf(input[row * NEURONS_LAYER3 + column]);
			atomicAdd(&sumOfRow, t);
			__syncthreads();

			output[row * NEURONS_LAYER3 + column] = t / sumOfRow;
		}
	}
}

void forwardWrapper_SoftMax(float* layer_input, float* output){
	dim3 kernelBlocks(BATCH_SIZE);
	dim3 kernelBlockThreads(NEURONS_LAYER3);
	forwardKernel_SoftMax <<<kernelBlocks, kernelBlockThreads>>> (layer_input, output);

	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "DeviceSync failed: " << cudaStatus;
		exit(EXIT_FAILURE);
	}
}