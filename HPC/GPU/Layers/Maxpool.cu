#include "hip/hip_runtime.h"
// #pragma once

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using std::cerr;
using std::cout;
using std::endl;

#include "../Network_Config.h"

__device__ float GetMax(float x1, float x2, float x3, float x4) {
	float tmp = x1;
	if (tmp < x2) tmp = x2;
	if (tmp < x3) tmp = x3;
	if (tmp < x4) tmp = x4;
	return tmp;
}

/// FORWARD ///

__global__ void forwardKernel_Maxpool(float* input, float* output, int* max_pool_idx, unsigned INPUT_IMG_DIM, unsigned INPUT_IMAGE_SIZE){
	const int i = blockIdx.x;
	const int row = threadIdx.x;

	const int nmbOfPictures = BATCH_SIZE * NMB_FILTERS;
	int nextBlock, nextRow, nextMatrix;
	int counter;

	if (i < nmbOfPictures) {
		if (row < INPUT_IMG_DIM && row % 2 == 0) {
			counter = i * (INPUT_IMG_DIM * INPUT_IMG_DIM / 4) + row * (INPUT_IMG_DIM / 4) + row / 2;
			for (int c = 0; c < INPUT_IMG_DIM; c += 2) {
				nextMatrix = i * INPUT_IMAGE_SIZE;
				nextBlock = c + row * INPUT_IMG_DIM + nextMatrix;
				nextRow = c + (row + 1) * INPUT_IMG_DIM + nextMatrix;

				float x1 = input[nextBlock];
				float x2 = input[nextBlock + 1];
				float x3 = input[nextRow];
				float x4 = input[nextRow + 1];

				output[counter] = GetMax(x1, x2, x3, x4);

				if (output[counter] == x1) max_pool_idx[counter] = nextBlock;
				if (output[counter] == x2) max_pool_idx[counter] = nextBlock + 1;
				if (output[counter] == x3) max_pool_idx[counter] = nextRow;
				if (output[counter] == x4) max_pool_idx[counter] = nextRow + 1;

				counter++;
			}
		}
	}
}

void forwardWrapper_Maxpool(float* input, float* output, int* max_pool_idx, unsigned INPUT_IMG_DIM, unsigned INPUT_IMAGE_SIZE){
	dim3 kernelBlocks(BATCH_SIZE * NMB_FILTERS);
	dim3 kernelBlockThreads(INPUT_IMG_DIM);
	forwardKernel_Maxpool <<<kernelBlocks, kernelBlockThreads>>> (input, output, max_pool_idx, INPUT_IMG_DIM, INPUT_IMAGE_SIZE);

	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "DeviceSync failed (Maxpool.cu -> forwardWrapper): " << cudaStatus;
		exit(EXIT_FAILURE);
	}
}


/// BACKWARD ///

__global__ void backwardKernel_Maxpool(float* input, float* output, int* max_pool_idx, unsigned output_image_size){
	const int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < BATCH_SIZE * NMB_FILTERS * output_image_size) {
		output[max_pool_idx[i]] = input[i];
	}
}

void backwardWrapper_Maxpool(float* input, float* output, int* max_pool_idx, unsigned output_image_size){
	hipError_t cudaStatus;
	cudaStatus = hipMemset(output, 0, BATCH_SIZE * NMB_FILTERS * CONV_OUTPUT_IMG_SIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		std::cerr << "hipMemset in backwardWrapper_Maxpool failed with " << cudaStatus << std::endl;
		exit(EXIT_FAILURE);
	}

	dim3 kernelBlocks(BATCH_SIZE * NMB_FILTERS);
	dim3 kernelBlockThreads(output_image_size);
	backwardKernel_Maxpool <<<kernelBlocks, kernelBlockThreads>>> (input, output, max_pool_idx, output_image_size);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "DeviceSync failed in backwardWrapper_Maxpool: " << cudaStatus;
		exit(EXIT_FAILURE);
	}
}