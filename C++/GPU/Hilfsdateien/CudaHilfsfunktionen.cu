#include "CudaHilfsfunktionen.h"

using std::cerr;
using std::cout;
using std::endl;

void cudaMalloc_Helper(float* object, const int nmbBytes, hipError_t &cudaStatus, const char* variableName){
	cudaStatus = hipMalloc((void **) & object, nmbBytes);
	cudaHelper(cudaStatus, true, variableName);
}


void cudaMemcpy_Helper(	float* dst,
						float* src,
						const int nmbBytes,
						const hipMemcpyKind mode,
						hipError_t &cudaStatus,
						const char* variableName){
	cudaStatus = hipMemcpy(dst, src, nmbBytes, mode);
	cudaHelper(cudaStatus, false, variableName);
}


// Wertet „cudaStatus“ aus
void cudaHelper(hipError_t &cudaStatus, const bool hipMalloc, const char* variableName){
	if(cudaStatus != hipSuccess) {
		std::cerr << "<" << (hipMalloc ? "hipMalloc" : "hipMemcpy") << "> ist fehlgeschlagen während <" << variableName << "> mit Fehlercode: " << cudaStatus << "!" << std::endl;
		std::cerr << "\t" << hipGetErrorString(cudaStatus) << std::endl;
		exit(EXIT_FAILURE);
	}
}


void printMat(float * matrix, const int nmbElements, const char* name){
	printf("%s = ", name);
	for(int i = 0; i < nmbElements; i++){
		printf("%.1f ", matrix[i]);
	}
	printf("\n");
}


void cudaDeviceSynchronize_Helper(hipError_t cudaStatus, const char* functionName){
	if(cudaStatus != hipSuccess){
		std::cerr << "hipDeviceSynchronize mit Status " << cudaStatus << " fehlgeschlagen bei " << functionName << std::endl;
		exit(EXIT_FAILURE);
	}
}