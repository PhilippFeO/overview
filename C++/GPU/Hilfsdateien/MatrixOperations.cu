#include "hip/hip_runtime.h"
#include "MatrixOperations.h"

using std::cout;
using std::endl;
using std::array;

// Für SoftMax-Abbildung
using std::max_element;
using std::exp;

/*
	Berechnet das Matrixprodukt A*B
*/
__global__ void matMul(float *A, float *B, float* result, const int rowsOfA, const int columnsOfA, const int columnsOfB){
	for (int r = 0; r < rowsOfA; r++) {
		for (int col = 0; col < columnsOfB; col++) {
			float res = 0.0f;
			for (int k = 0; k < columnsOfA; k++) {
				res += A[r * columnsOfA + k] * B[k * columnsOfB + col];
			}
			result[r * columnsOfB + col] = res;
		}
	}
}

__global__ void matMul_parallel(float *A, float *B, float* result, const int rowsOfA, const int columnsOfA, const int columnsOfB){
	/*
		Die Ausgabe-Matrix hat bspw r*c-Elemente. Für jedes dieser wird ein Thread gestartet.
		Da man pro Block maximal 1024 Threads nutzen kann, müssen ggfl. weitere Blöcke hinzugenommen werden.
		In diesem Fall muss der Index des Threads erst aus den Block- & Thread-Koordinaten berechnet werden.
	*/
	int row = blockIdx.x;
	int column = blockIdx.y * MAX_THREADS + threadIdx.x;
	if (row < rowsOfA && column < columnsOfB) {
		result[row * columnsOfB + column] = 0;
		for (int i = 0; i < columnsOfA; i++) {
			result[row * columnsOfB + column] += A[row * columnsOfA + i] * B[i * columnsOfB + column];
		}
	}
}

/*
	Transponiere eine Matrix

	Template-Parameter:
	ROWS:		Zeilen der Matrix
	COLUMNS:	Spalten der Matrix
*/
__global__ void transpose(float* matrix, float* transposedMatrix, const int ROWS, const int COLUMNS){
	for (int row = 0; row < ROWS; row++) {
		for (int column = 0; column < COLUMNS; column++) {
			transposedMatrix[column * ROWS + row] = matrix[row * COLUMNS + column];
		}
	}
}
__global__ void transpose_parallel(float *matrix, float *transposedMatrix, const int ROWS, const int COLUMNS){
	// const int row = blockIdx.x;
	// const int column = threadIdx.x;
	// transposedMatrix[column * ROWS + row] = matrix[row * COLUMNS + column];	
	const int row = blockIdx.x;
	const int column = blockIdx.y * MAX_THREADS + threadIdx.x;	// Jede Reihe wird in "blockIdx.y" 1024 Elemente-lange Sequenzen unterteilt (außer evtl. der Rest)
	if(row < ROWS && column < COLUMNS){	// Der Index muss innerhalb einer Reihe liegen, dh. die berechnete Spalte darf nicht größer als die der Matrix sein
		transposedMatrix[column * ROWS + row] = matrix[row * COLUMNS + column];
	}
}


/*
	Skalare Multiplikation

	Template-Parameter:
	nmbElements: Anzahl der Elemente beider Matrizen

	Funktionsparameter:
	scalar: float -- Skalar mit dem multipliziert werden soll
	matrix: array<float, nmbElements> -- Matrix, deren Einträge multipliziert werden sollen

	Rückgabewert:
	result: array<float, nmbElements> -- Die skalarmultiplizierte Matrix
*/
__global__ void scalarMult(const float scalar, float *matrix, const int ROWS, const int COLUMNS) {
	for (int i = 0; i < ROWS * COLUMNS; i++) {
		matrix[i] = scalar * matrix[i];
	}
}
__global__ void scalarMult_parallel(const float scalar, float *matrix, const int ROWS, const int COLUMNS) {
	const int row = blockIdx.x;
	const int column = threadIdx.x;
	matrix[row * COLUMNS + column] = scalar * matrix[row * COLUMNS + column];
}


/*
	Berechnet die Differenz zweier Matrizen

	Template-Parameter:
	rows:		Zeilen der Matrix
	columns:	Spalten der Matrix
*/
__global__ void matMinus(float *left, float *right, const int ROWS, const int COLUMNS) {
	for (int i = 0; i < ROWS * COLUMNS; i++) {
		left[i] = left[i] - right[i];
	}
}
__global__ void matMinus_parallel(float *left, float *right, const int ROWS, const int COLUMNS) {
	const int row = blockIdx.x;
	const int column = threadIdx.x;
	const int idx = row * COLUMNS + column;
	left[idx] = left[idx] - right[idx];
}


__global__ void printMatrix(float *matrix, const int rows, const int columns){
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < columns; c++) {
			float tmp = matrix[r * columns + c];
			// Da y_hat in SoftMax überschrieben wird, steht dort durch das Label eine negative Zahl.
			// Dies wird hier für die Ausgabe korrigiert.
			printf("%.4f ", (tmp < 0 ? tmp + 1 : tmp));
		}
		printf("\n");
	}
}

void printMatrix2(float *matrix, const int rows, const int columns){
	for (int r = 0; r < rows; r++) {
		for (int c = 0; c < columns; c++) {
			float tmp = matrix[r * columns + c];
			// Da y_hat in SoftMax überschrieben wird, steht dort durch das Label eine negative Zahl.
			// Dies wird hier für die Ausgabe korrigiert.
			printf("%.4f ", (tmp < 0 ? tmp + 1 : tmp));
		}
		printf("\n");
	}
}