//Cuda
#include "hip/hip_runtime.h"
#include ""
// Schichten inkludieren
#include "./Layers/ReLU.h"
#include "./Layers/Fully.h"
#include "./Layers/L2_Loss.h"
#include "./Layers/SoftMax.h"
#include "./Layers/Maxpool.h"
#include "./Layers/Conv.h"

#include "Network_Config.h"

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <chrono>

using std::cerr;
using std::vector;
using std::string;
using std::ifstream;

vector<string> split(const string &s, char delim) {
	std::stringstream ss(s);
	string item;
	vector<string> tokens;
	while (getline(ss, item, delim)) {
		tokens.push_back(item);
	}
	return tokens;
}

int main(){

	printf("+++ %d %d %d\n", ITERATIONS, BATCH_SIZE, NMB_FILTERS);

	string line;
    vector<string> line_v;

    // cout << "Loading data ...\n";
    // x_train_NN wird alle Bilder linear enthalten
    vector<float> x_train_NN;
    vector<float> y_train_NN;

	//Size of y is 42000×1, and the size of X is 42000×784. 
	//Every line of X is a 28×28 grayscale picture of a handwritten number. 
	//Every element of y is a number from 0 to 9.
    ifstream myfile ("train.txt");
    if (myfile.is_open())
    {
		int ii = 0;
        while ( getline (myfile,line) )
        {
            line_v = split(line, '\t');
            float digit = strtof((line_v[0]).c_str(),0);
            for (unsigned i = 0; i < 10; ++i) {
                if (i == digit)
                {
                    y_train_NN.push_back(1.);
                }
                else y_train_NN.push_back(0.);
            }
            
            unsigned size = static_cast<unsigned>(line_v.size());
            for (unsigned i = 1; i < size; ++i) {
                x_train_NN.push_back(strtof((line_v[i]).c_str(),0)/255.0f);
            }
        }
        myfile.close();
    }

	// Grafikkarte auswählen
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
		exit(EXIT_FAILURE);
	}

	// Speicher für Trainingsdaten auf der GPU allokieren
	float* x_train;
	cudaStatus = hipMalloc((void**)&x_train, x_train_NN.size() * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* y_train;
	cudaStatus = hipMalloc((void**)&y_train, y_train_NN.size() * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	// Trainingsdaten auf GPU kopieren
	cudaStatus = hipMemcpy(x_train, &x_train_NN[0], x_train_NN.size() * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cerr << "hipMemcpy failed!";
		exit(EXIT_FAILURE);
	}

	cudaStatus = hipMemcpy(y_train, &y_train_NN[0], y_train_NN.size() * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cerr << "hipMemcpy failed!";
		exit(EXIT_FAILURE);
	}

	// Initialisieren der Schichten
    Conv<NMB_FILTERS, FILTER_DIM, TRAIN_IMAGE_DIM> Conv;
    Maxpool<CONV_OUTPUT_IMG_DIM, MAXPOOL_OUTPUT_IMG_DIM, NMB_FILTERS> Maxpool;
    Fully<NEURONS_LAYER1, NEURONS_LAYER2> FC_12;
    ReLU<BATCH_SIZE, NEURONS_LAYER2> ReLU_2;
    Fully<NEURONS_LAYER2, NEURONS_LAYER3> FC_23;    
    SoftMax SoftMax;
    L2_Loss L2_Loss;

    // Datenstrukturen für die Vorwärtspropagation allokieren
	float* Conv_output;
	cudaStatus = hipMalloc((void**)&Conv_output, BATCH_SIZE * NMB_FILTERS * CONV_OUTPUT_IMG_SIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* Maxpool_output;
	cudaStatus = hipMalloc((void**)&Maxpool_output, BATCH_SIZE * NMB_FILTERS * MAXPOOL_OUTPUT_IMG_SIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* FC_12_output;
	cudaStatus = hipMalloc((void**)&FC_12_output, BATCH_SIZE * NEURONS_LAYER2 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* ReLU_2_output;
	cudaStatus = hipMalloc((void**)&ReLU_2_output, BATCH_SIZE * NEURONS_LAYER2 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* FC_23_output;
	cudaStatus = hipMalloc((void**)&FC_23_output, BATCH_SIZE * NEURONS_LAYER3 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* y_hat;
	cudaStatus = hipMalloc((void**)&y_hat, BATCH_SIZE * NEURONS_LAYER3 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

    float loss;

	float *loss_gpu;
	cudaStatus = hipMalloc((void **)&loss_gpu, sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

    // Datenstrukturen für die Rückwärtspropagation allokieren
	float* d_y_hat;
	cudaStatus = hipMalloc((void**)&d_y_hat, BATCH_SIZE * NEURONS_LAYER3 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* FC_23_backward_output;
	cudaStatus = hipMalloc((void**)&FC_23_backward_output, BATCH_SIZE * NEURONS_LAYER2 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* ReLU_2_backward_output;
	cudaStatus = hipMalloc((void**)&ReLU_2_backward_output, BATCH_SIZE * NEURONS_LAYER2 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* FC_12_backward_output;
	cudaStatus = hipMalloc((void**)&FC_12_backward_output, BATCH_SIZE * NEURONS_LAYER1 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	float* Maxpool_backward_output;
	cudaStatus = hipMalloc((void**)&Maxpool_backward_output, BATCH_SIZE * NMB_FILTERS * CONV_OUTPUT_IMG_SIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}
	
	float* Conv_backward_output;
	cudaStatus = hipMalloc((void**)&Conv_backward_output, BATCH_SIZE * TRAIN_IMAGE_SIZE * sizeof(float));
	if (cudaStatus != hipSuccess) {
		cerr << "hipMalloc failed!";
		exit(EXIT_FAILURE);
	}

	// cout << "Training the model ...\n";

	// const int iterations = 100;

	auto begin = std::chrono::high_resolution_clock::now();

    for (unsigned i = 0; i < ITERATIONS; ++i) {
        // Building batches of input variables (x) and labels (y)
        // unsigned indx = (i * BATCH_SIZE) % (2000-BATCH_SIZE);
        int indx = rand() % (42000-BATCH_SIZE);


        // Vorwärtspropagation
		Conv.forward(&x_train[indx*TRAIN_IMAGE_SIZE], Conv_output);
		Maxpool.forward(Conv_output, Maxpool_output);		
		FC_12.forward(Maxpool_output, FC_12_output);
		ReLU_2.forward(FC_12_output, ReLU_2_output);
		FC_23.forward(ReLU_2_output, FC_23_output);
		SoftMax.forward(FC_23_output, y_hat);
		L2_Loss.forward(y_hat, &y_train[indx*10], loss_gpu);

		//loss_gpu auf cpu kopieren
		cudaStatus = hipMemcpy(&loss, loss_gpu, sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			cerr << "hipMemcpy failed!";
			exit(EXIT_FAILURE);
		}

        // Rückwärtspropagation
		L2_Loss.backward(y_hat, &y_train[indx*10], d_y_hat);
		FC_23.backward(d_y_hat, FC_23_backward_output);
		ReLU_2.backward(FC_23_backward_output, ReLU_2_backward_output);
		FC_12.backward(ReLU_2_backward_output, FC_12_backward_output);
		Maxpool.backward(FC_12_backward_output, Maxpool_backward_output);
		Conv.backward(Maxpool_backward_output, Conv_backward_output);

		// cout << "[GPU] Epoch " << i << "   Loss: " << 2 * loss / BATCH_SIZE << endl;

        //if ((i+1) % 100 == 0){
        //    // cout << "Epoch " << i+1 << endl;
        //    cout << "-----------------------------------------------Epoch " << i+1 << "--------------------------------------------------" <<"\n";
        //    cout << "Predictions:" << "\n";
        //    //printMatrix<BATCH_SIZE, NEURONS_LAYER3>(y_hat, 10, 10);
        //    cout << "Ground truth:" << "\n";
        //    //printMatrix<BATCH_SIZE, NEURONS_LAYER3>(b_y, 10, 10);
        //    cout << "                                            Loss " << 2*loss/BATCH_SIZE <<"\n";
        //    cout << "--------------------------------------------End of Epoch :(------------------------------------------------" <<"\n";
        //};
	}

	auto end = std::chrono::high_resolution_clock::now();

	std::ofstream file;
	file.open("../time_measurements/GPU_Par_" + std::to_string(ITERATIONS) + "-Iter_" + std::to_string(BATCH_SIZE) + "-Batchsize_" + std::to_string(NMB_FILTERS) + "-Filters.txt", std::ios_base::app);

	auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end-begin).count();
	file << duration << "\n";

	// for(int i = 0; i < 11; i++){
	// 	file << arrr[i] << "\n";
	// }
	// file << ">>> NEW MEASUREMENT <<<\n";

	file.close();

	// Speicher freigeben
	hipFree(x_train);
	hipFree(y_train);

	hipFree(Conv_output);
	hipFree(Maxpool_output);
	hipFree(FC_12_output);
	hipFree(ReLU_2_output);
	hipFree(FC_23_output);
	hipFree(y_hat);
	hipFree(loss_gpu);

	hipFree(d_y_hat);
	hipFree(FC_23_backward_output);
	hipFree(ReLU_2_backward_output);
	hipFree(FC_12_backward_output);
	hipFree(Maxpool_backward_output);
	hipFree(Conv_backward_output);

	return 0;
}