
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <string>

#include "../Network_Config.h"
#include "../Hilfsdateien/CudaHilfsfunktionen.h"
#include "../Hilfsdateien/MatrixOperations.h"

// Berechnet die Koordiaten-Darstellung für die zu startenden Blöcke

dim3 calc_blocks(const int threads){
    const int rest = threads % MAX_THREADS == 0 ? 0 : 1;
    const int blocks_per_row = threads / MAX_THREADS + rest;
    //printf("columns / MAX_THREADS + rest = %d\n", blocks_per_row);
    const dim3 blocks_for_transpose(BATCH_SIZE, blocks_per_row);
    return blocks_for_transpose;
}

void printToFile_parallel(float *data_GPU, const int nmbElements, std::string filename, const bool logging){
	if (logging) {
		float* data_CPU = new float[nmbElements];
		hipMemcpy(data_CPU, data_GPU, nmbElements * sizeof(float), hipMemcpyDeviceToHost);
		std::ofstream out(("./Ausgaben/parallel_" + filename + ".txt").c_str());
		for (int i = 0; i < nmbElements; i++) {
			// out<<"["<<i<<"] "<<data_CPU[i]<<"\n";
			out << data_CPU[i] << "\n";
		}
		out.close();
		delete[] data_CPU;
	}
}

/// FORWARD ///////////////////////////////////////////

void forwardWrapper_Fully(float* layer_input, float* W, float* output, const int NEURONS_IN, const int NEURONS_OUT){
	// Führe Kernel-Funktion aus
	// Jeder Thread übernimmt eine Reihen-Spalte-Multiplikation
	// Es gibt BATCH_SIZE * NEURONS_OUT viele solcher Multiplikationen
	// => BATCH_SIZE * NEURONS_OUT Threads
	dim3 nmbBlocks(BATCH_SIZE);	// Ein Block für jede Reihe
	dim3 threadsPerBlock(NEURONS_OUT); // Ein Thread für jede Spalte 
	matMul_parallel<<<nmbBlocks, threadsPerBlock>>>(layer_input, W, output, BATCH_SIZE, NEURONS_IN, NEURONS_OUT);
	// printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
	// 	nmbBlocks.x, nmbBlocks.y, nmbBlocks.z,
	// 	threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
	// matMul<<<1, 1>>>(layer_input, W, output, BATCH_SIZE, NEURONS_IN, NEURONS_OUT);
	cudaDeviceSynchronize_Helper(hipDeviceSynchronize(), "matMul (Fully, Vorwärtsschritt)");
}



/// BACKWARD //////////////////////////////////////////

void backwardWrapper_Fully(	float* error_tensor,
							float* W,
							float* d_input, 
							float* layer_input,
							const int NEURONS_IN,
							const int NEURONS_OUT){

	const bool logging = true;
	
	float *W_transposed;
	const int nmbBytesOfW_transposed = NEURONS_IN * NEURONS_OUT * sizeof(float);
	hipError_t cudaStatus;

	// Reserviere Speicher für „W_transposed“
	cudaStatus = hipMalloc((void **) &W_transposed, nmbBytesOfW_transposed);
	cudaHelper(cudaStatus, true, "W_transposed");

	// Rückwärtsschritt: error_tensor * W.T
	// Transponiere Matrix „W“ steht nun in „W_transposed“ und befindet sich bereits auf der GPU
	// Für jede Reihe einen Block und für jede Spalte einen Thread
	transpose_parallel<<< NEURONS_IN, NEURONS_OUT >>>(W, W_transposed, NEURONS_IN, NEURONS_OUT);
	// transpose<<<1, 1>>>(W, W_transposed, BATCH_SIZE, NEURONS_IN);
	cudaDeviceSynchronize_Helper(hipDeviceSynchronize(), "transpose 1 (Fully parallel, Rückwärtsschritt)");
		// printToFile_parallel(W_transposed, NEURONS_IN * NEURONS_OUT, "W_transposed", logging);

	// Führe Rückwärtsschritt durch
	// matMul<<<1, 1>>>(error_tensor, W_transposed, d_input, BATCH_SIZE, NEURONS_IN, NEURONS_OUT);
	const dim3 blocks_for_matMul = calc_blocks (NEURONS_IN);
	matMul_parallel<<< blocks_for_matMul, MAX_THREADS >>> (error_tensor, W_transposed, d_input, BATCH_SIZE, NEURONS_OUT, NEURONS_IN);
	// matMul_parallel<<< BATCH_SIZE, NEURONS_IN >>> (error_tensor, W_transposed, d_input, BATCH_SIZE, NEURONS_OUT, NEURONS_IN);
	cudaDeviceSynchronize_Helper(hipDeviceSynchronize(), "matMul 1 (Fully parallel, Rückwärtsschritt)");
		// printToFile_parallel(d_input, BATCH_SIZE * NEURONS_IN, "d_input", logging);


	// Gradientenabstieg: W = W - learning_rate * d_W * error_tensor
	// Ableitung bzgl Gewichte: d_W = input.T * _
	float *d_W;
	const int nmbBytes_d_W = BATCH_SIZE * NEURONS_IN * sizeof(float);
	cudaStatus = hipMalloc((void **) &d_W, nmbBytes_d_W);
	cudaHelper(cudaStatus, true, "d_W");
	// transpose<<<1, 1>>>(layer_input, d_W, BATCH_SIZE, NEURONS_IN);
	dim3 blocks_for_transpose = calc_blocks(NEURONS_IN);
	transpose_parallel<<< blocks_for_transpose, MAX_THREADS >>>(layer_input, d_W, BATCH_SIZE, NEURONS_IN);
	// transpose_parallel<<< BATCH_SIZE, NEURONS_IN >>>(layer_input, d_W, BATCH_SIZE, NEURONS_IN);
	cudaDeviceSynchronize_Helper(hipDeviceSynchronize(), "transpose 2 (Fully parallel, Gradientenabstieg)");
		// printToFile_parallel(layer_input, BATCH_SIZE * NEURONS_IN, "input_transposed", logging);

	// Berechne d_W * error_tensor = input.T * error_tensor
	float *tmp;
	const int nmbBytes_tmp = NEURONS_IN * NEURONS_OUT * sizeof(float);
	cudaStatus = hipMalloc((void **) &tmp, nmbBytes_tmp);
	cudaHelper(cudaStatus, true, "tmp");
	// matMul<<<1, 1>>>(d_W, error_tensor, tmp, NEURONS_IN, BATCH_SIZE, NEURONS_OUT);
	matMul_parallel<<< NEURONS_IN, NEURONS_OUT >>>(d_W, error_tensor, tmp, NEURONS_IN, BATCH_SIZE, NEURONS_OUT);
	cudaDeviceSynchronize_Helper(hipDeviceSynchronize(), "matMul 2 (Fully parallel, Gradientenabstieg)");
		// printToFile_parallel(tmp, NEURONS_IN * NEURONS_OUT, "d_W_times_error_tensor", logging);

	// W = W - learning_rate * (d_W * error_tensor) = W - learning_rate * tmp
	// scalarMult<<<1, 1>>>(learning_rate, tmp, NEURONS_IN, NEURONS_OUT);
	scalarMult_parallel<<< NEURONS_IN, NEURONS_OUT >>>(
		learning_rate,
		tmp,
		NEURONS_IN, NEURONS_OUT);
	cudaDeviceSynchronize_Helper(hipDeviceSynchronize(), "scalarMult (Fully, Gradientenabstieg)");
	
	// matMinus<<<1, 1>>>(W, tmp, NEURONS_IN, NEURONS_OUT);
	matMinus_parallel<<< NEURONS_IN, NEURONS_OUT >>> (W, tmp, NEURONS_IN, NEURONS_OUT);
	cudaDeviceSynchronize_Helper(hipDeviceSynchronize(), "matMinus (Fully, Gradientenabstieg)");
		// printToFile_parallel(W, NEURONS_IN * NEURONS_OUT, "neue_Gewichte", logging);

	// Speicher freigeben
	hipFree(W_transposed); hipFree(d_W); hipFree(tmp);
}