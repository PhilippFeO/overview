#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include "../Network_Config.h"

using std::cerr;

#define FILTER_SIZE FILTER_DIM * FILTER_DIM
#define INPUT_IMG_SIZE INPUT_IMG_DIM * INPUT_IMG_DIM
#define OUTPUT_IMG_DIM (INPUT_IMG_DIM - FILTER_DIM + 1)
#define OUTPUT_IMG_SIZE OUTPUT_IMG_DIM * OUTPUT_IMG_DIM

__global__ void forwardKernel(float* input, float* output, float* filters, unsigned filter_num, unsigned filter_dim, unsigned input_img_dim) {
	// �ber alle Bilder iterieren
	const unsigned i = blockIdx.x;
	// �ber alle Filter iterieren
	const unsigned f = blockIdx.y;
	// �ber Zeilen des Bildes iterieren
	const unsigned r = threadIdx.x;
	// �ber Spalten des Bildes iterieren
	const unsigned c = threadIdx.y;
	float tmp = 0;
	// �ber Zeilen des Filter iterieren
	for (unsigned filter_r = 0; filter_r < filter_dim; ++filter_r) {
		// �ber Spalten des Filters iterieren
		for (unsigned filter_c = 0; filter_c < filter_dim; ++filter_c) {
			tmp += input[input_img_dim * input_img_dim * i + (r + filter_r) * input_img_dim + c + filter_c]
				* filters[f * filter_dim * filter_dim + filter_r*filter_dim + filter_c];
		}
	}
	output[i * (input_img_dim - filter_dim + 1) * (input_img_dim - filter_dim + 1) * filter_num
		+ f * (input_img_dim - filter_dim + 1) * (input_img_dim - filter_dim + 1)
		+ r * (input_img_dim - filter_dim + 1) + c] = tmp;
}

void forwardWrapper(float * input, float * output, float* filters, unsigned filter_num, unsigned filter_dim, unsigned input_img_dim)
{
	dim3 numBlocks(BATCH_SIZE, filter_num);
	dim3 threadsPerBlock((input_img_dim - filter_dim + 1), (input_img_dim - filter_dim + 1));
	forwardKernel << <numBlocks, threadsPerBlock >> > (input, output, filters, filter_num, filter_dim, input_img_dim);
	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "DeviceSync failed:" << cudaStatus;
		exit(EXIT_FAILURE);
	}
}

__global__ void backwardKernel(float* error_tensor, float* d_input, float* filters, float* d_filters, float* layer_input,
	unsigned FILTERS, unsigned FILTER_DIM, unsigned INPUT_IMG_DIM) {

	// �ber alle Bilder iterieren
	//for (int layer_input_image_index = 0; layer_input_image_index < BATCH_SIZE; ++layer_input_image_index) {
	const int layer_input_image_index = blockIdx.x;
	const int filter_index = threadIdx.x;
	const int rr = threadIdx.y;
	const int cc = threadIdx.z;
	d_filters[layer_input_image_index * FILTER_SIZE * FILTERS + filter_index * FILTER_SIZE + FILTER_DIM * rr + cc] = 0;
	// �ber die Zeilen des Bildes iterieren
	for (int r = 0; r < OUTPUT_IMG_DIM; ++r) {
		// �ber die Spalten des Bildes iterieren
		for (int c = 0; c < OUTPUT_IMG_DIM; ++c) {
			// �ber alle Filter iterieren
			//for (int filter_index = 0; filter_index < FILTERS; ++filter_index) {
			// �ber Zeilen des Filters iterieren
			//for (int rr = 0; rr < FILTER_DIM; ++rr) {
				// �ber Spalten des Filters iterieren
				//for (int cc = 0; cc < FILTER_DIM; ++cc) {
					// Gradient bzgl. der Gewichte
			d_filters[layer_input_image_index * FILTER_SIZE * FILTERS + filter_index * FILTER_SIZE + FILTER_DIM * rr + cc] +=
				error_tensor[layer_input_image_index * FILTERS * OUTPUT_IMG_SIZE + filter_index * OUTPUT_IMG_SIZE + r * OUTPUT_IMG_DIM + c] *
				layer_input[INPUT_IMG_SIZE * layer_input_image_index + (r + rr) * INPUT_IMG_DIM + (c + cc)];
			// Gradient bez�glich der Eingabe
			//d_input[layer_input_image_index * INPUT_IMG_SIZE + (r + rr) * INPUT_IMG_DIM + (c + cc)] +=
			//	error_tensor[layer_input_image_index * FILTERS * OUTPUT_IMG_SIZE + filter_index * OUTPUT_IMG_SIZE + r * OUTPUT_IMG_DIM + c] *
			//	filters[filter_index * FILTER_SIZE + rr * FILTER_DIM + cc];
		//}
	//}
	//}
		}
	}
	//}
}

__global__ void backwardInit(float* d_input, float* d_filters, unsigned FILTERS) {
	// Ableitungsdatenstruktur mit 0 initialisieren
	for (int i = threadIdx.x; i < BATCH_SIZE * TRAIN_IMAGE_SIZE; i += 32) d_input[i] = 0;
}

__global__ void gradient(float* filters, float* d_filters, unsigned FILTERS, const float learning_rate) {
	// Gradientenabstieg: Filter - learning_rate * d_Filter
	for (int i = threadIdx.x; i < FILTERS * FILTER_SIZE; i += 32) {
		filters[i] -= learning_rate * d_filters[i];
	}
}

__global__ void reduktion(float* g_odata, float* d_input, const unsigned FILTERS, const unsigned FILTER_DIM) {
	extern __shared__ float sdata[];
	if (threadIdx.x < BATCH_SIZE) {
		sdata[threadIdx.x] = g_odata[threadIdx.x * FILTER_SIZE * FILTERS + blockIdx.x * FILTER_SIZE + blockIdx.y * FILTER_DIM + blockIdx.z];
	}
	else {
		sdata[threadIdx.x] = 0;
	}
	__syncthreads();

	for (int s = 1; s < blockDim.x; s *= 2) {
		if (threadIdx.x % (2 * s) == 0) {
			//printf("%d %d %d\n", threadIdx.x, s, threadIdx.x + s);
			sdata[threadIdx.x] += sdata[threadIdx.x + s];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) d_input[blockIdx.x * FILTER_SIZE + blockIdx.y * FILTER_DIM + blockIdx.z] = sdata[0];
}

int reduktionThreads() {
	if (!((BATCH_SIZE & (BATCH_SIZE - 1)) == 0)) {
		if (BATCH_SIZE < 4) return 4;
		if (BATCH_SIZE < 8) return 8;
		if (BATCH_SIZE < 16) return 16;
		if (BATCH_SIZE < 32) return 32;
		if (BATCH_SIZE < 64) return 64;
		if (BATCH_SIZE < 128) return 128;
		if (BATCH_SIZE < 256) return 256;
		if (BATCH_SIZE < 512) return 512;
		if (BATCH_SIZE < 1024) return 1024;
	}
	else {
		return BATCH_SIZE;
	}
}

void backwardWrapper(float* error_tensor, float* d_input, float* filters, float* d_filters, float* layer_input,
	unsigned FILTERS, unsigned FILTER_DIM, unsigned INPUT_IMG_DIM, float* g_odata)
{
	backwardInit << <1, 32 >> > (d_input, d_filters, FILTERS);
	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "DeviceSync failed:" << cudaStatus;
		exit(EXIT_FAILURE);
	}

	dim3 threadsPerBlock(FILTERS, FILTER_DIM, FILTER_DIM);
	backwardKernel << <BATCH_SIZE, threadsPerBlock >> > (error_tensor, d_input, filters, g_odata, layer_input,
		FILTERS, FILTER_DIM, INPUT_IMG_DIM);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "DeviceSync failed:" << cudaStatus;
		exit(EXIT_FAILURE);
	}

	//Reduktion
	int threads = reduktionThreads();
	reduktion << <threadsPerBlock, threads, threads * sizeof(float) >> > (g_odata, d_filters, FILTERS, FILTER_DIM);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "DeviceSync failed:" << cudaStatus;
		exit(EXIT_FAILURE);
	}


	gradient << <1, 32 >> > (filters, d_filters, FILTERS, learning_rate);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "DeviceSync failed:" << cudaStatus;
		exit(EXIT_FAILURE);
	}
}
