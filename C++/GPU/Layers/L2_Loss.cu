#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using std::cerr;
using std::cout;
using std::endl;

#include "../Network_Config.h"

/*
	Berechnet die Differenz zweier Matrizen

	Template-Parameter:
	rows:		Zeilen der Matrix
	columns:	Spalten der Matrix
*/
__global__ void matMinus_L2_Loss(float* left, float* right) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < BATCH_SIZE * NEURONS_LAYER3) {
		left[idx] = left[idx] - right[idx];
	}

}


/// FORWARD ///

__global__ void forwardKernel_L2_Loss(float* y_hat, float* labels, float* output) {
	const int row = blockIdx.x;
	const int column = threadIdx.x;

	float tmp = 0.5 * y_hat[row * NEURONS_LAYER3 + column] * y_hat[row * NEURONS_LAYER3 + column];
	atomicAdd(output, tmp);
}

__global__ void forwardInit_L2_Loss(float* output) {
	*output = 0;
}


void forwardWrapper_L2_Loss(float* y_hat, float* labels, float* output) {
	forwardInit_L2_Loss << <1, 1 >> > (output);
	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "[forwardWrapper_L2_Loss -> matMinus] DeviceSync failed: " << cudaStatus << std::endl;
		exit(EXIT_FAILURE);
	}

	dim3 kernelBlocks(BATCH_SIZE);
	dim3 kernelBlockThreads(NEURONS_LAYER3);

	matMinus_L2_Loss << <kernelBlocks, kernelBlockThreads >> > (y_hat, labels);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "[forwardWrapper_L2_Loss -> matMinus] DeviceSync failed: " << cudaStatus << std::endl;
		exit(EXIT_FAILURE);
	}

	forwardKernel_L2_Loss << <kernelBlocks, kernelBlockThreads >> > (y_hat, labels, output);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "[forwardWrapper_L2_Loss -> forwardKernel] DeviceSync failed: " << cudaStatus << std::endl;
		exit(EXIT_FAILURE);
	}
}


/// BACKWARD ///

__global__ void backwardKernel_L2_Loss(float* y_hat, float* labels, float* output) {
	// Kann man mehr oder weniger überspringen, weil „x - y“, also „y_hat - labels“ schon im Vorwärtsschritt
	// und in „y_hat“ gespeichert wurde
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < BATCH_SIZE * NEURONS_LAYER3) {
		output[idx] = y_hat[idx];
	}
}

void backwardWrapper_L2_Loss(float* y_hat, float* labels, float* output) {
	dim3 kernelBlocks(BATCH_SIZE);
	dim3 kernelBlockThreads(NEURONS_LAYER3);
	backwardKernel_L2_Loss << <kernelBlocks, kernelBlockThreads >> > (y_hat, labels, output);

	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "[backwardWrapper_L2_Loss] DeviceSync failed: " << cudaStatus;
		exit(EXIT_FAILURE);
	}
}