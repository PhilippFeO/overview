#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using std::cerr;
using std::cout;
using std::endl;

#include<cmath>
using std::ceil;
using std::floor;

#include "../Network_Config.h"
#include "../Hilfsdateien/CudaHilfsfunktionen.h"

#define MAX_THREADS 1024
#define MAX_THREADS_IN_X 32

/// FORWARD ///

__global__ void forwardKernel_ReLU(float* input, float* output, const int ROWS, const int COLUMNS){
	// const int idx = (threadIdx.x + threadIdx.y * MAX_THREADS_IN_X) + blockIdx.x * blockDim.x * blockDim.y;
	// if(idx < ROWS * COLUMNS){
		// output[idx] = input[idx] <= 0 ? 0 : input[idx];
	// }
	const int row = blockIdx.x;
	const int column = threadIdx.x;
	const int idx = row * COLUMNS + column;
	output[idx] = input[idx] <= 0 ? 0 : input[idx];
}

void forwardWrapper_ReLU(float* layer_input, float *output, const int ROWS, const int COLUMNS){
	// const int blocks = ceil((ROWS * COLUMNS) / MAX_THREADS);
		// dim3 threadsPerBlock(COLUMNS);
		// dim3 numBlocks(ROWS);
		// printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
		// 	numBlocks.x, numBlocks.y, numBlocks.z,
		// 	threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
	forwardKernel_ReLU<<<ROWS, COLUMNS>>>(layer_input, output, ROWS, COLUMNS);
	cudaDeviceSynchronize_Helper(hipDeviceSynchronize(), "forwardKernel_ReLU");
}


/// BACKWARD ///

__global__ void backwardKernel_ReLU(float* error_tensor, float *layer_input, float *d_input, const int ROWS, const int COLUMNS){
	// const int idx = (threadIdx.x + threadIdx.y * MAX_THREADS_IN_X) + blockIdx.x * blockDim.x * blockDim.y;
	// if(idx < ROWS * COLUMNS){
	// 	error_tensor[idx] = layer_input[idx] <= 0 ? 0 : error_tensor[idx];
	// }
	const int row = blockIdx.x;
	const int column = threadIdx.x;
	const int idx = row * COLUMNS + column;
	d_input[idx] = layer_input[idx] <= 0 ? 0 : error_tensor[idx];
}

void backwardWrapper_ReLU(float *error_tensor, float *layer_input, float *d_input, const int ROWS, const int COLUMNS){	
	// const int blocks = ceil((ROWS * COLUMNS) / MAX_THREADS);
	// dim3 threadsPerBlock(MAX_THREADS_IN_X, MAX_THREADS_IN_X);
	// dim3 numBlocks(blocks);
	// backwardKernel_ReLU<<<numBlocks, threadsPerBlock>>>(error_tensor, layer_input, d_input, ROWS, COLUMNS);
	backwardKernel_ReLU<<<ROWS, COLUMNS>>>(error_tensor, layer_input, d_input, ROWS, COLUMNS);
	cudaDeviceSynchronize_Helper(hipDeviceSynchronize(), "backwardKernel_ReLU");
}